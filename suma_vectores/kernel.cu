#include "hip/hip_runtime.h"
/*
	Daniel Finestrat Martinez, 48719584H.
	Vamos a pasar de hacer 1 operacion con CPU a 25600 o m�s al mismo tiempo con GPU
	
	Para cambiar el tamanyo del vector cambiar kNumElements
	Para cambiar el numero maximo de hilos por bloque usar maxThreadsPerBlock
	Para cambiar el numero maximo de bloques por grid usar maxBlocksPerGrid

	CUDA Device Query en "C:\ProgramData\NVIDIA Corporation\CUDA Samples\v8.0\1_Utilities\deviceQuery"
	960 Max Threads per Block = 1024
	960 Max Blocks per Grid = 65536 (Usamos 65535)

	CPU = HOST
	GPU = DEVICE
	Kernel = isntrucciones de GPU que se ejecuta en un millon o dos millones de copias diferentes a la vez
	__global__ = Se ejecuta en GPU pero se llama desde CPU
*/

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <Windows.h>

// Retorna (a - b) en segundos
double performancecounter_diff(LARGE_INTEGER *a, LARGE_INTEGER *b) {
	LARGE_INTEGER freq;
	QueryPerformanceFrequency(&freq);
	return (double)(a->QuadPart - b->QuadPart) / (double)freq.QuadPart;
}

//Suma de vectores de mucho tamanyo, sea el numero elementos divisible entre 2 o no
__global__ void kernel_suma_vectores(const float * cpA, const float * cpB, float * cpC, const int kNumElem) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int salto = blockDim.x * gridDim.x; //Metemos un salto para que cada hilo ejecute mas de una operacion 
	for ( ; i < kNumElem; i += salto) { cpC[i] = cpA[i] + cpB[i]; }
}

int main(){

	LARGE_INTEGER t_ini, t_fin;	//Tiempo inicial y final de Proceso
	double secsH; //Tiempo de operacion

	//PASO 1: INICIALIZACION
	hipSetDevice(0); //Llamamos a cualquier variable del kernel para inicializar, esta nos dice q usemos la GPU 0
	
	//PASO 2: DECLARACION
	//Apuntamos el tamanyo maximo de bloque y de grid de nuestra grafica
	std::cerr << "Creamos los Vectores.\n\n";

	const int maxThreadsPerBlock = 1024; //Ajustar esta cifra segun tu GPU
	const int maxBlocksPerGrid = 65535; //Ajustar esta cifra segun tu GPU



	const int kNumElements = 90000000; //Apuntamos numero de elementos de cada vector que sumaremos (ej. 25600)
	size_t vector_size_bytes = kNumElements * sizeof(float); //Obtenemos cantidad de memoria necesaria para un vector del numero de elementos definido, siendo variables float

	//Reservamos vector_size_bytes de memoria para cada vector que vayamos a usar em la CPU (Host)
	float *h_A_ = (float *)malloc(vector_size_bytes);
	float *h_B_ = (float *)malloc(vector_size_bytes);
	float *h_C_ = (float *)malloc(vector_size_bytes);
	if (h_A_ == NULL || h_B_ == NULL || h_C_ == NULL) { std::cerr << ("MEMORIA MAL RESERVADA"); getchar(); exit(-1); } //Comprobamos errores

	//Ahora declaramos los mismos pero en la grafica y reservamos la memoria
	float *d_A_ = NULL; float *d_B_ = NULL; float *d_C_ = NULL;
	hipMalloc((void **)&d_A_, vector_size_bytes); //Puntero a puntero para reservar memoria?
	hipMalloc((void **)&d_B_, vector_size_bytes);
	hipMalloc((void **)&d_C_, vector_size_bytes);

	std::cerr << "Rellenamos los Vectores.\n\n";
	//Inicializamos los vectores a numeros aleatorios
	for (int i = 0; i < kNumElements; ++i) { //++i mas eficiente que i++
		h_A_[i] = rand() / (float)RAND_MAX;
		h_B_[i] = rand() / (float)RAND_MAX;
	}

	//PASO 3: TRANSFERENCIA
	std::cerr << "Transferimos los datos al Device.\n\n";
	hipMemcpy(d_A_, h_A_, vector_size_bytes, hipMemcpyHostToDevice); //Copia de CPU a GPU
	hipMemcpy(d_B_, h_B_, vector_size_bytes, hipMemcpyHostToDevice);

	//PASO 4: EJECUCION
	std::cerr << "Empezamos la Ejecucion\n\n";
	int threadsPerBlock =  maxThreadsPerBlock; //Ponemos el tamanyo al maximo para minimizar los bloques necesarios

	/*
	Redondeamos el numero de bloques por grid HACIA ARRIBA para evitar desbordamiento con vectores arbitrarios.
	Para ello usamos la formula z = 1 + ((x - 1) / y) que automaticamente nos sube el numero INTEGRER al siguiente
	cuando nos pasamos. De este modo:
		1 + ((25600 - 1) / 256) = 100.99 = 100 (ya que INT trunca)
		1 + ((25601 - 1) / 256) = 101
	*/
	int blocksPerGrid = 1 + ((kNumElements - 1) / threadsPerBlock);
	if (blocksPerGrid > maxBlocksPerGrid) { blocksPerGrid = maxBlocksPerGrid; } //Si blocksPerGrid es mayor que el que podemos usar, lo dividimos para obtener el maximo numero de bloques por grid admitido

	dim3 block(threadsPerBlock, 1, 1); //Definimos los bloques, x sera los hilos que contiene cada bloque
	dim3 grid(blocksPerGrid, 1, 1); //Definimos la grid con 3 variables, x sera los bloques que contiene cada grid
	
	kernel_suma_vectores<<<grid, block>>> (d_A_, d_B_, d_C_, kNumElements); //Invocacion de kernel. Le pasamos variables de DISPOSITIVO (GPU)

	//Comprobamos si tenemos errores
	hipError_t err_ = hipGetLastError();
	if (err_ != hipSuccess) {
		std::cerr << hipGetErrorString(err_);
		getchar(); exit(-1);
	}

	//PASO 5: TRANSFERENCIA
	hipMemcpy(h_C_, d_C_, vector_size_bytes, hipMemcpyDeviceToHost); //Nos traemos los datos a CPU desde GPU

	//Comprobamos si nos da igual (CON UMBRAL de 1e-5)

	QueryPerformanceCounter(&t_ini);
	for (int i = 0; i < kNumElements; ++i) {
		if (fabs(h_A_[i] + h_B_[i] - h_C_[i] > 1e-5)) {
			std::cerr << "Error de verificacion en posicion " << i << ".";
			getchar(); exit(-1);
		}
	}
	QueryPerformanceCounter(&t_fin);
	secsH = performancecounter_diff(&t_fin, &t_ini);

	//PASO 6: LIBERAR MEMORIA Y DISPOSITIVO
	free(h_A_); free(h_B_); free(h_C_);
	hipFree(d_A_); hipFree(d_B_); hipFree(d_C_);
	hipDeviceReset();
	
	int numOperacionesPorThread = 1 + ((kNumElements - 1) / (threadsPerBlock * blocksPerGrid));
	std::cerr << "Operaciones realizadas correctamente.\n\n";
	printf("Hemos sumado vectores de %d posiciones, usando bloques de %d hilos (el maximo permitido por nuestra grafica)", kNumElements, threadsPerBlock);
	printf(" y grids de %d bloques (el tamanyo maximo de grid de nuestra grafica es de %d bloques), por lo tanto cada thread", blocksPerGrid, maxBlocksPerGrid);
	printf(" realizara un maximo de %d operacion(es).", numOperacionesPorThread);
	printf("\n\nSe tardaron %f segundos en ejecutar la operacion en el Host", secsH);
	getchar();

	return 0;
}